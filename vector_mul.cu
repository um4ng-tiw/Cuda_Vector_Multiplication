
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 1000000


//Kernel to perform the multiplication of array elements
__global__ void vect_mul(int*a, int *b, int*c)
{

    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < N)
        c[tid]=a[tid]*b[tid];
    
}

void vect_mul_cpu(int* a, int* b, int* c)
{
    for(int i=0;i<N;i++)
    {
        a[i]=b[i]*c[i];
    }
}


int* initialize(int* ptr)
{
    for(int i=0;i<N;i++)
    {
        ptr[i]=rand() % 10000;
    }

    return ptr;
}

int main()
{
    //Array declaration and initialization for CPU side
    int *a = (int*)malloc(N*sizeof(int));
    int *b = (int*)malloc(N*sizeof(int));

    //Initializing array with random values
    a=initialize(a);
    b=initialize(b);


    int* c = (int*)malloc(N*sizeof(int)); //For cpu execution

    int *ptr_c_cpu =(int*)malloc(N*sizeof(int)); //For copying value from gpu to cpu

    int *ptr_a_cpu = a;
    int *ptr_b_cpu = b;


    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    
    //Declaring pointer variables for the GPU
    int *ptr_a_gpu;
    int *ptr_b_gpu;
    int *ptr_c_gpu;

    //Allocating memory to pointer variables in the GPU
    hipMalloc((void **)&ptr_a_gpu, N * sizeof(int));
    hipMalloc((void **)&ptr_b_gpu, N * sizeof(int));
    hipMalloc((void **)&ptr_c_gpu, N * sizeof(int));

    //Copying contents of variable from cpu(host) to gpu(device)
    hipMemcpy(ptr_a_gpu, ptr_a_cpu, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(ptr_b_gpu, ptr_b_cpu, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(ptr_c_gpu, ptr_c_cpu, N * sizeof(int), hipMemcpyHostToDevice);

    //Calling the kernel to perform execution on gpu
    hipEventRecord(start);
    vect_mul<<<1000,1000>>>(ptr_a_gpu,ptr_b_gpu,ptr_c_gpu);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    //Copying the result array from gpu(device) to cpu(host)
    hipMemcpy(ptr_c_cpu,ptr_c_gpu,N*sizeof(int),hipMemcpyDeviceToHost);
    

    float time_taken_gpu = 0.0;
    hipEventElapsedTime(&time_taken_gpu, start, stop);


    clock_t t;
    t=clock();
    vect_mul_cpu(a,b,c); //Calling the function for cpu execution
    t=clock()-t;
    double time_taken_cpu = ((double)t)/CLOCKS_PER_SEC; //Recorded in seconds
    time_taken_cpu*=1000.0;


    printf("Time taken by CPU:%f ms\n",time_taken_cpu);
    printf("Time taken by GPU:%f ms\n",time_taken_gpu);
  

    //Freeing space in host memory
    free(ptr_c_cpu);
    free(a);
    free(b);
    free(c);

    //Freeing space in gpu
    hipFree(ptr_a_gpu);
    hipFree(ptr_b_gpu);
    hipFree(ptr_c_gpu);
}


